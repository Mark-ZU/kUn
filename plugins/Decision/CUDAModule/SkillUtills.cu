﻿#include <hip/hip_runtime.h>


#include "hip/device_functions.h"
#include <math_functions.h>
#include <math.h>
#include <stdio.h>

#define FRAME_PERIOD (1 / 60.0)
#define ZERO_NUM (1e-8)
#define A_FACTOR (1.5)
#define OUR_MAX_ACC (450)
#define OUR_MAX_DEC (450)
#define OUR_MAX_VEL (300)
#define THEIR_MAX_ACC (500)
#define THEIR_MAX_DEC (500)
#define THEIR_MAC_VEL (350)
#define PI (3.14159265359)
#define G (9.8)
#define SQRT_2 (1.414)
#define TIME_FOR_OUR (0)
#define TIME_FOR_OUR_BOTH_KEEP (-0.2)
#define TIME_FOR_THEIR_BOTH_KEEP (-0.4)
#define TIME_FOR_THEIR (-0.6)
#define TIME_FOR_JUDGE_HOLDING (0.5)

#define FRICTION (87)
#define PLAYER_CENTER_TO_BALL_CENTER (60)
#define MAX_PLAYER_NUM (12)
#define THREAD_NUM (128)
#define BLOCK_X (16)
#define BLOCK_Y (MAX_PLAYER_NUM * 2)
#define MAX_BALL_SPEED (650)
#define MIN_BALL_SPEED (50)
#define BALL_SPEED_UNIT ((MAX_BALL_SPEED - MIN_BALL_SPEED) / BLOCK_X)
#define MIN_DELTA_TIME (0.2)

#define MAX_CHIP_SPEED (650)
#define MIN_CHIP_SPEED (100)
#define CHIP_SPEED_UNIT ((MAX_CHIP_SPEED - MIN_CHIP_SPEED) / BLOCK_X)

namespace  {
    const int FIELD_LENGTH = 1200;
    const int FIELD_WIDTH  = 900;
    const int PENALTY_LENGTH = 120;
    const int PENALTY_WIDTH = 240;
}

typedef struct {
    double x, y;
} Vector;

typedef struct {
    double x, y;
} Point;

typedef struct {
    Point Pos;
    Vector Vel;
    bool isValid;
} Player;

typedef struct {
    Point interPos;
    double interTime;
    double Vel;
    float dir;
    int playerIndex;
    double deltaTime;
    double Q;
} rType;

__device__ bool if_finite(double a) {
    return fabs(a) < 9999;
}

__device__ bool IsInField(Point p, double buffer=10) {
    return (p.x > -FIELD_LENGTH / 2 + buffer && p.x < FIELD_LENGTH / 2 - buffer
            && p.y < FIELD_WIDTH / 2 - buffer && p.y > -FIELD_WIDTH / 2 + buffer);
}

__device__ bool IsInPenalty(Point p, double buffer=20) {
    return (p.x < -FIELD_LENGTH/2 + PENALTY_LENGTH && p.x > -FIELD_LENGTH/2 && p.y > -PENALTY_WIDTH/2 && p.y < PENALTY_WIDTH/2)
            || (p.x > FIELD_LENGTH/2 - PENALTY_LENGTH && p.x < FIELD_LENGTH/2 && p.y > -PENALTY_WIDTH/2 && p.y < PENALTY_WIDTH/2);
}

__device__ void CUDA_compute_motion_1d(double x0, double v0, double v1,
    double a_max, double d_max, double v_max, double a_factor,
    double &traj_accel, double &traj_time, double &traj_time_acc, double &traj_time_dec, double &traj_time_flat)
{
    if (x0 == 0 && v0 == v1) {
        traj_accel = 0;
        traj_time_acc = traj_time_dec = 0;
        return;
    }

    if (!if_finite(x0) || !if_finite(v0) || !if_finite(v1)) {
        traj_accel = 0;
        traj_time_acc = traj_time_dec = 0;
        return;
    }

    a_max /= a_factor;
    d_max /= a_factor;

    double accel_time_to_v1 = fabs(v1 - v0) / a_max;
    double accel_dist_to_v1 = fabs((v1 + v0) / 2.0) * accel_time_to_v1;
    double decel_time_to_v1 = fabs(v0 - v1) / d_max;
    double decel_dist_to_v1 = fabs((v0 + v1) / 2.0) * decel_time_to_v1;

    double period = 1 / 40.0;

    if (v0 * x0 > 0 || (fabs(v0) > fabs(v1) && decel_dist_to_v1 > fabs(x0))) {
        double time_to_stop = fabs(v0) / (d_max);
        double x_to_stop = v0 * v0 / (2.0 * d_max);

        CUDA_compute_motion_1d(x0 + copysign(x_to_stop, v0), 0, v1, a_max * a_factor, d_max * a_factor,
            v_max, a_factor, traj_accel, traj_time, traj_time_acc, traj_time_dec, traj_time_flat);
        traj_time += time_to_stop;
        traj_time /= 1.25;
        traj_accel = 0;

        return;
    }

    if (fabs(v0) > fabs(v1)) {
        traj_time_acc = (sqrt((d_max * v0 * v0 + a_max * (v1 * v1 + 2 * d_max * fabs(x0))) / (a_max + d_max)) - fabs(v0)) / a_max;

        if (traj_time_acc < 0.0)
            traj_time_acc = 0;
        traj_time_dec = ((fabs(v0) - fabs(v1)) + a_max * traj_time_acc) / d_max;
    }

    else if (accel_dist_to_v1 > fabs(x0)) {
        traj_time_acc = (sqrt(v0 * v0 + 2 * a_max * fabs(x0)) - fabs(v0)) / a_max;
        traj_time_dec = 0.0;
    }

    else {
        traj_time_acc = (sqrt((d_max * v0 * v0 + a_max * (v1 * v1 + 2 * d_max * fabs(x0))) / (a_max + d_max)) - fabs(v0)) / a_max;
        if (traj_time_acc < 0.0)
            traj_time_acc = 0;
        traj_time_dec = ((fabs(v0) - fabs(v1)) + a_max * traj_time_acc) / d_max;
    }


    if (traj_time_acc * a_max + fabs(v0) > v_max) {
        double dist_without_flat = (v_max * v_max - v0 * v0) / (2 * a_max) + (v_max * v_max - v1 * v1) / (2 * d_max);
        traj_time_flat = (fabs(x0) - dist_without_flat) / v_max;
    }
    else {
        traj_time_flat = 0;
    }


    double a_to_v1_at_x0 = fabs(v0 * v0 - v1 * v1) / (2 * fabs(x0));
    double t_to_v1_at_x0 = (-fabs(v0) + sqrt(v0 * v0 + 2 * fabs(a_to_v1_at_x0) * fabs(x0))) / fabs(a_to_v1_at_x0);
    if (t_to_v1_at_x0 < period) {
        traj_accel = -copysign(a_to_v1_at_x0, v0);
        return;
    }

    if (FRAME_PERIOD * a_max + fabs(v0) > v_max && traj_time_flat > period) {
        traj_time = traj_time_flat + traj_time_dec;
        traj_accel = 0;
    }
    else if (traj_time_acc < period && traj_time_dec == 0.0) {
        traj_time = traj_time_acc;
        traj_accel = copysign(a_max * a_factor, -x0);
    }
    else if (traj_time_acc < period && traj_time_dec > 0.0) {
        traj_time = traj_time_dec;
        traj_accel = copysign(d_max * a_factor, -v0);

    }
    else {
        traj_time = traj_time_acc + traj_time_flat / 1.1 + traj_time_dec / 1.1;
        traj_accel = copysign(a_max * a_factor, -x0);
    }
}

__device__ double CUDA_predictedTime(double x0, double y0, double x1, double y1, double vx, double vy) {
    double timeX, timeXAcc, timeXDec, timeXFlat, acc;
    double timeY, timeYAcc, timeYDec, timeYFlat;
    double x = x0 - x1;
    double y = y0 - y1;
    double newVelAngle = atan2(vy, vx) - atan2(y, x);
    double length = sqrt(vx * vx + vy * vy);
    x = sqrt(x * x + y * y);
    y = 0.0;
    vx = length * cospi(newVelAngle);
    vy = length * sinpi(newVelAngle);
    CUDA_compute_motion_1d(x, vx, 0, 450, 450, 300, 1.5, acc, timeX, timeXAcc, timeXDec, timeXFlat);
    CUDA_compute_motion_1d(y, vy, 0, 450, 450, 300, 1.5, acc, timeY, timeYAcc, timeYDec, timeYFlat);
    if (timeX < 1e-5 || timeX > 50) timeX = 0;
    if (timeY < 1e-5 || timeY > 50) timeY = 0;
    return (timeX > timeY ? timeX : timeY);
}

__device__ double CUDA_predictedTheirTime(double x0, double y0, double x1, double y1, double vx, double vy) {
    double timeX, timeXAcc, timeXDec, timeXFlat, acc;
    double timeY, timeYAcc, timeYDec, timeYFlat;
    double x = x0 - x1;
    double y = y0 - y1;
    double newVelAngle = atan2(vy, vx) - atan2(y, x);
    double length = sqrt(vx * vx + vy * vy);
    x = sqrt(x * x + y * y);
    y = 0.0;
    vx = length * cospi(newVelAngle);
    vy = length * sinpi(newVelAngle);
    CUDA_compute_motion_1d(x, vx, 0, 500, 500, 350, 1.5, acc, timeX, timeXAcc, timeXDec, timeXFlat);
    CUDA_compute_motion_1d(y, vy, 0, 500, 500, 350, 1.5, acc, timeY, timeYAcc, timeYDec, timeYFlat);
    if (timeX < 1e-5 || timeX > 50) timeX = 0;
    if (timeY < 1e-5 || timeY > 50) timeY = 0;
    return (timeX > timeY ? timeX : timeY);
}

__device__ bool CUDA_predictedInterTime(Point mePoint, Point ballPoint, Vector meVel, Vector ballVel, Point* interceptPoint, double* interTime, double responseTime) {
    if(sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y) < 40){
        *interceptPoint = ballPoint;
        *interTime = CUDA_predictedTime(mePoint.x, mePoint.y, interceptPoint->x, interceptPoint->y, meVel.x, meVel.y);
        return true;
    }
    const double ballAcc = FRICTION / 2;
    double ballArriveTime = 0;
    double meArriveTime = 9999;
    const double stepTime = 0.1;
    double testBallLength = 0;
    Point testPoint = ballPoint;
    double testVel = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
    double max_time = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y) / ballAcc;

    bool canInter = true;
    for (ballArriveTime = 0; ballArriveTime < max_time; ballArriveTime += stepTime ) {
        testVel = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y) - ballAcc*ballArriveTime;//v_0-at
        testBallLength = PLAYER_CENTER_TO_BALL_CENTER + (sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y) + testVel)*ballArriveTime / 2;
        Vector direc;
        direc.x = testBallLength * ballVel.x / sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
        direc.y = testBallLength * ballVel.y / sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
        testPoint.x = ballPoint.x + direc.x;
        testPoint.y = ballPoint.y + direc.y;
        meArriveTime = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y);
        if(meArriveTime < 0.15) meArriveTime = 0;
        if(IsInPenalty(testPoint)) continue;
        if (!IsInField(testPoint)) {
            canInter = false;
            break;
        }
        if(meArriveTime + responseTime < ballArriveTime) break;
    }
    if(!canInter || ballArriveTime >= max_time) {
        interceptPoint->x = 9999;
        interceptPoint->y = 9999;
        *interTime = 9999;
        return false;
    }
    *interceptPoint = testPoint;
    *interTime = CUDA_predictedTime(mePoint.x, mePoint.y, interceptPoint->x, interceptPoint->y, meVel.x, meVel.y);
    return true;
}

__device__ bool CUDA_predictedChipInterTime(Point mePoint, Point ballPoint, Vector meVel, Vector ballVel, Point* interceptPoint, double* interTime, double responseTime) {
    double chipVel = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
    double meArriveTime = 9999;
    double ballAcc = FRICTION / 2.0;
    double stepTime = 0.1;
    double testBallLength = 0;
    Point testPoint = ballPoint;

    double factor_1 = 0.2;
    double factor_2 = 0.1;

    double time_1 = SQRT_2 * chipVel / 100.0 / G;
    double time_2 = SQRT_2 * chipVel * factor_1 / 100.0 / G;

    double length_1 = chipVel * time_1 / SQRT_2;
    double length_2 = chipVel * time_2 * factor_1 / SQRT_2;


    double moveVel = chipVel / SQRT_2 * factor_2;

    bool canInter = true;
    double ballDropTime = time_1 + time_2;
    double max_time = moveVel / ballAcc + ballDropTime;
    double afterArrivedTime = 0;
    while (afterArrivedTime < max_time) {

        Vector direc;
        testBallLength = length_1 + length_2 + (moveVel * afterArrivedTime - 0.5 * ballAcc * afterArrivedTime * afterArrivedTime);

        direc.x = testBallLength * ballVel.x / sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
        direc.y = testBallLength * ballVel.y / sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
        testPoint.x = ballPoint.x + direc.x;
        testPoint.y = ballPoint.y + direc.y;
        meArriveTime = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y);
        if(meArriveTime < 0.10) meArriveTime = 0;

        if(IsInPenalty(testPoint)) {
            afterArrivedTime += stepTime;
            continue;
        }
        if (!IsInField(testPoint)) {
            canInter = false;
            break;
        }
        if(meArriveTime + responseTime < ballDropTime + afterArrivedTime) break;
        afterArrivedTime += stepTime;
    }

    if(!canInter || afterArrivedTime >= max_time) {
        interceptPoint->x = 9999;
        interceptPoint->y = 9999;
        *interTime = 9999;
        return false;
    }

    *interceptPoint = testPoint;
    *interTime = CUDA_predictedTime(mePoint.x, mePoint.y, interceptPoint->x, interceptPoint->y, meVel.x, meVel.y);
    return true;
}

__device__ bool CUDA_predictedTheirInterTime(Point mePoint, Point ballPoint, Vector meVel, Vector ballVel, Point* interceptPoint, double* interTime, double responseTime) {
    if(sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y) < 40){
        *interceptPoint = ballPoint;
        *interTime = CUDA_predictedTheirTime(mePoint.x, mePoint.y, interceptPoint->x, interceptPoint->y, meVel.x, meVel.y);
        return true;
    }
    const double ballAcc = FRICTION / 2;
    double ballArriveTime = 0;
    double meArriveTime = 9999;
    const double stepTime = 0.1;
    double testBallLength = 0;
    Point testPoint = ballPoint;
    double testVel = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
    double max_time = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y) / ballAcc;

    bool canInter = true;
    for (ballArriveTime = 0; ballArriveTime < max_time; ballArriveTime += stepTime ) {
        testVel = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y) - ballAcc*ballArriveTime;//v_0-at
        testBallLength = PLAYER_CENTER_TO_BALL_CENTER + (sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y) + testVel)*ballArriveTime / 2;
        Vector direc;
        direc.x = testBallLength * ballVel.x / sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
        direc.y = testBallLength * ballVel.y / sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
        testPoint.x = ballPoint.x + direc.x;
        testPoint.y = ballPoint.y + direc.y;
        meArriveTime = CUDA_predictedTheirTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y);
        if(meArriveTime < 0.15) meArriveTime = 0;
        if(IsInPenalty(testPoint)) continue;
        if (!IsInField(testPoint)) {
            canInter = false;
            break;
        }
        if(meArriveTime + responseTime < ballArriveTime) break;
    }
    if(!canInter) {
        interceptPoint->x = 9999;
        interceptPoint->y = 9999;
        *interTime = 9999;
        return false;
    }
    *interceptPoint = testPoint;
    *interTime = CUDA_predictedTime(mePoint.x, mePoint.y, interceptPoint->x, interceptPoint->y, meVel.x, meVel.y);
    return true;
}

__global__ void calculateAllInterInfo(Player* players, Point* ballPos, rType* bestPass) {
    int angleIndex = threadIdx.x;
    int speedIndex = blockIdx.x;
    int playerNum =  blockIdx.y;

    Vector ballVel;
    ballVel.x = (speedIndex * BALL_SPEED_UNIT + MIN_BALL_SPEED) * cospi(2*PI* angleIndex / THREAD_NUM);
    ballVel.y = (speedIndex * BALL_SPEED_UNIT + MIN_BALL_SPEED) * sinpi(2*PI* angleIndex / THREAD_NUM);

    double interTime;
    Point interPoint;
    interTime = 9999;
    interPoint.x = 9999;
    interPoint.y = 9999;
    if( players[playerNum].isValid && playerNum < 12)
         CUDA_predictedInterTime(players[playerNum].Pos, *ballPos, players[playerNum].Vel, ballVel, &interPoint, &interTime, 0);
    else if(players[playerNum].isValid)
         CUDA_predictedTheirInterTime(players[playerNum].Pos, *ballPos, players[playerNum].Vel, ballVel, &interPoint, &interTime, 0);

    int offset = blockIdx.y + gridDim.y * (threadIdx.x + blockIdx.x * blockDim.x);
    bestPass[offset].interPos = interPoint;
    bestPass[offset].interTime = interTime;
    bestPass[offset].playerIndex = playerNum;
    bestPass[offset].dir = 2*PI* angleIndex / THREAD_NUM;
    bestPass[offset].Vel = speedIndex * BALL_SPEED_UNIT + MIN_BALL_SPEED;
//    /***************** chip *******************/
    interTime = 9999;
    interPoint.x = 9999;
    interPoint.y = 9999;
    ballVel.x = (speedIndex * CHIP_SPEED_UNIT + MIN_CHIP_SPEED) * cospi(2 * PI * angleIndex / THREAD_NUM);
    ballVel.y = (speedIndex * CHIP_SPEED_UNIT + MIN_CHIP_SPEED) * sinpi(2 * PI * angleIndex / THREAD_NUM);

    if( players[playerNum].isValid && playerNum < 12)
         CUDA_predictedChipInterTime(players[playerNum].Pos, *ballPos, players[playerNum].Vel, ballVel, &interPoint, &interTime, 0);
      else if(players[playerNum].isValid)
         CUDA_predictedChipInterTime(players[playerNum].Pos, *ballPos, players[playerNum].Vel, ballVel, &interPoint, &interTime, 0);


    offset += BLOCK_X * BLOCK_Y * THREAD_NUM;
    bestPass[offset].interPos = interPoint;
    bestPass[offset].interTime = interTime;
    bestPass[offset].playerIndex = playerNum;
    bestPass[offset].dir = 2 * PI * angleIndex / THREAD_NUM;
    bestPass[offset].Vel = speedIndex * CHIP_SPEED_UNIT + MIN_CHIP_SPEED;
    __syncthreads();
}

__global__ void getBest(rType* passPoints) {
    __shared__ rType iP[BLOCK_Y];
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int playerNum = threadIdx.x;
    iP[playerNum] = passPoints[blockId * blockDim.x + playerNum];
    __syncthreads();
    bool even = true;
    for(int i = 0; i < blockDim.x; i++) {
        if(playerNum < blockDim.x - 1 && even && iP[playerNum].interTime > iP[playerNum + 1].interTime) {
            rType temp;
            temp = iP[playerNum + 1];
            iP[playerNum + 1] = iP[playerNum];
            iP[playerNum] = temp;
        }
        else if(playerNum > 0 && !even && iP[playerNum].interTime < iP[playerNum - 1].interTime) {
            rType temp;
            temp = iP[playerNum];
            iP[playerNum] = iP[playerNum - 1];
            iP[playerNum - 1] = temp;
        }
        even = !even;
        __syncthreads();
    }
    passPoints[blockId * blockDim.x + playerNum] = iP[playerNum];

    /************************/
    __shared__ rType iP2[BLOCK_Y];
    iP2[playerNum] = passPoints[blockId * blockDim.x + playerNum + BLOCK_X * BLOCK_Y * THREAD_NUM];
    __syncthreads();
    even = true;
    for(int i = 0; i < blockDim.x; i++) {
        if(playerNum < blockDim.x - 1 && even && iP2[playerNum].interTime > iP2[playerNum + 1].interTime) {
            rType temp;
            temp = iP2[playerNum + 1];
            iP2[playerNum + 1] = iP2[playerNum];
            iP2[playerNum] = temp;
        }
        else if(playerNum > 0 && !even && iP2[playerNum].interTime < iP2[playerNum - 1].interTime) {
            rType temp;
            temp = iP2[playerNum];
            iP2[playerNum] = iP2[playerNum - 1];
            iP2[playerNum - 1] = temp;
        }
        even = !even;
        __syncthreads();
    }
    passPoints[blockId * blockDim.x + playerNum + BLOCK_X * BLOCK_Y * THREAD_NUM] = iP2[playerNum];
    __syncthreads();
}


extern "C" void BestPass(Player* players, Point* ball, rType* result) {
    rType *bestPass;

    hipMallocManaged((void**)&bestPass, 2 * BLOCK_X * BLOCK_Y * THREAD_NUM * sizeof(rType));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dim3 bolcks(BLOCK_X, BLOCK_Y);
    calculateAllInterInfo <<< bolcks, THREAD_NUM >>> (players, ball, bestPass);

    dim3 blocks2(BLOCK_X, THREAD_NUM);
    getBest<<< blocks2, BLOCK_Y >>> (bestPass);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess){
        printf("CUDA ERROR: %d\n", (int)cudaStatus);
        printf("Error Name: %s\n", hipGetErrorName(cudaStatus));
        printf("Description: %s\n", hipGetErrorString(cudaStatus));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %.5f ms\n", milliseconds);

    rType defaultPlayer;
    defaultPlayer.dir = 9999;
    defaultPlayer.interPos.x = 9999;
    defaultPlayer.interPos.y = 9999;
    defaultPlayer.interTime = 9999;
    defaultPlayer.Vel = 9999;
    defaultPlayer.deltaTime = -9999;
    for(int i = 0; i < BLOCK_X * BLOCK_Y * THREAD_NUM; i += BLOCK_Y) {
        int playerNum = 0;
        for(int j = 0; j < MAX_PLAYER_NUM; j++) {
            if(bestPass[i + j].playerIndex > 11) {
                while(playerNum < MAX_PLAYER_NUM) {
                    result[i / 2 + playerNum] = defaultPlayer;
                    playerNum++;
                }
                for(int k = 0; k < j; k++) {
                    result[i / 2 + k].deltaTime = bestPass[i + j].interTime - result[i / 2 + k].interTime;
                    if(result[i / 2 + k].deltaTime < MIN_DELTA_TIME)
                        result[i / 2 + k] = defaultPlayer;
                }
                break;
            }
            else {
                result[i / 2 + playerNum] = bestPass[i + j];
                playerNum++;
            }
        }
    }
    for(int i = BLOCK_X * BLOCK_Y * THREAD_NUM; i < 2 * BLOCK_X * BLOCK_Y * THREAD_NUM; i += BLOCK_Y) {
        int playerNum = 0;

        for(int j = 0; j < MAX_PLAYER_NUM; j++) {

            if(bestPass[i + j].playerIndex > 11) {
                while(playerNum < MAX_PLAYER_NUM) {
                    result[i / 2 + playerNum] = defaultPlayer;
                    playerNum++;
                }
                for(int k = 0; k < j; k++) {
                    result[i / 2 + k].deltaTime = bestPass[i + j].interTime - result[i / 2 + k].interTime;
                    if(result[i / 2 + k].deltaTime < MIN_DELTA_TIME)
                        result[i / 2 + k] = defaultPlayer;
                }
                break;
            }
            else {
                result[i / 2 + playerNum] = bestPass[i + j];
                playerNum++;
            }
        }
    }
    hipFree(bestPass);
}

